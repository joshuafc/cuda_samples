#include "hip/hip_runtime.h"
#include <iostream>
#include "opencv2/opencv.hpp"


#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <chrono>

inline std::shared_ptr<void> MyCudaMalloc(int size)
{
    void* tmp = nullptr;
    if( hipError_t::hipSuccess != hipMalloc(&tmp, size))
        throw std::bad_alloc();
    return {tmp, [](void* p){ hipFree(p); }};
}

#define TILE_K 16
#define TILE_M 128
#define TILE_N 128
#define TILE_M_4 32
#define TILE_N_4 32

// A B C are all row-major
__global__ void MatrixMul_T_T(int cTileK, int m, int n, int k, float alpha,  const float* A, int lda, const float* B, int ldb, float beta,  float* C, int ldc)
{
    float4 f4_zero = make_float4(0.f, 0.f, 0.f, 0.f);
    float4 result[4][4] = {{f4_zero } };
    float4 reg_a[2];
    float4 reg_b[2];

    __shared__ float4 smemA[2][TILE_K][TILE_M_4];
    __shared__ float4 smemB[2][TILE_K][TILE_N_4];

    int tx_4, ty_4, tx_32, ty_32;
    {
        int ltid = threadIdx.y * blockDim.x + threadIdx.x;
        tx_4 = ltid % 4;
        ty_4 = ltid / 4;
        tx_32 = ltid % 32;
        ty_32 = ltid / 32;
    }

    int tileIdx=0;
    float4 ldA[2], ldB[2];
    {
        const float* pTileA = A + blockIdx.y * TILE_M * lda + tileIdx * TILE_K;
        const float* pTileB = B + tileIdx * TILE_K * ldb + blockIdx.x * TILE_N;
        ldA[0] = *(float4*)( pTileA + ty_4 * lda + tx_4 * 4 );
        ldA[1] = *(float4*)( pTileA + (ty_4 + 64) * lda + tx_4 * 4 );
        ldB[0] = *(float4*)( pTileB + ty_32 * ldb + tx_32 * 4 );
        ldB[1] = *(float4*)( pTileB + (ty_32 + 8) * ldb + tx_32 * 4 );

        *((float*)&smemA[0][0][0] + (tx_4*4    ) * TILE_M + ty_4 ) = ldA[0].x;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 1) * TILE_M + ty_4 ) = ldA[0].y;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 2) * TILE_M + ty_4 ) = ldA[0].z;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 3) * TILE_M + ty_4 ) = ldA[0].w;

        *((float*)&smemA[0][0][0] + (tx_4*4    ) * TILE_M + ty_4 + 64) = ldA[1].x;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 1) * TILE_M + ty_4 + 64) = ldA[1].y;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 2) * TILE_M + ty_4 + 64) = ldA[1].z;
        *((float*)&smemA[0][0][0] + (tx_4*4 + 3) * TILE_M + ty_4 + 64) = ldA[1].w;

        smemB[0][ty_32  ][tx_32] = ldB[0];
        smemB[0][ty_32+8][tx_32] = ldB[1];
    }

    __syncthreads();
    int write_stage_idx = 1;
    do
    {
        int load_stage_idx = write_stage_idx ^ 1;
        ++tileIdx;
        if( tileIdx < cTileK )
        {
            const float* pTileA = A + blockIdx.y * TILE_M * lda + tileIdx * TILE_K;
            const float* pTileB = B + tileIdx * TILE_K * ldb + blockIdx.x * TILE_N;
            ldA[0] = *(float4*)( pTileA + ty_4 * lda + tx_4 * 4 );
            ldA[1] = *(float4*)( pTileA + (ty_4 + 64) * lda + tx_4 * 4 );
            ldB[0] = *(float4*)( pTileB + ty_32 * ldb + tx_32 * 4 );
            ldB[1] = *(float4*)( pTileB + (ty_32 + 8) * ldb + tx_32 * 4 );
        }

#pragma unroll
        for(int subTileIdx=0; subTileIdx < TILE_K; ++subTileIdx)
        {
            reg_a[0] = smemA[load_stage_idx][subTileIdx][threadIdx.y];
            reg_a[1] = smemA[load_stage_idx][subTileIdx][threadIdx.y+16];

            reg_b[0] = smemB[load_stage_idx][subTileIdx][threadIdx.x];
            reg_b[1] = smemB[load_stage_idx][subTileIdx][threadIdx.x+16];

            result[0][0].x += reg_a[0].x * reg_b[0].x;
            result[0][0].y += reg_a[0].x * reg_b[0].y;
            result[0][0].z += reg_a[0].x * reg_b[0].z;
            result[0][0].w += reg_a[0].x * reg_b[0].w;
            result[0][1].x += reg_a[0].y * reg_b[0].x;
            result[0][1].y += reg_a[0].y * reg_b[0].y;
            result[0][1].z += reg_a[0].y * reg_b[0].z;
            result[0][1].w += reg_a[0].y * reg_b[0].w;
            result[0][2].x += reg_a[0].z * reg_b[0].x;
            result[0][2].y += reg_a[0].z * reg_b[0].y;
            result[0][2].z += reg_a[0].z * reg_b[0].z;
            result[0][2].w += reg_a[0].z * reg_b[0].w;
            result[0][3].x += reg_a[0].w * reg_b[0].x;
            result[0][3].y += reg_a[0].w * reg_b[0].y;
            result[0][3].z += reg_a[0].w * reg_b[0].z;
            result[0][3].w += reg_a[0].w * reg_b[0].w;

            // ----------------------

            result[1][0].x += reg_a[0].x * reg_b[1].x;
            result[1][0].y += reg_a[0].x * reg_b[1].y;
            result[1][0].z += reg_a[0].x * reg_b[1].z;
            result[1][0].w += reg_a[0].x * reg_b[1].w;
            result[1][1].x += reg_a[0].y * reg_b[1].x;
            result[1][1].y += reg_a[0].y * reg_b[1].y;
            result[1][1].z += reg_a[0].y * reg_b[1].z;
            result[1][1].w += reg_a[0].y * reg_b[1].w;
            result[1][2].x += reg_a[0].z * reg_b[1].x;
            result[1][2].y += reg_a[0].z * reg_b[1].y;
            result[1][2].z += reg_a[0].z * reg_b[1].z;
            result[1][2].w += reg_a[0].z * reg_b[1].w;
            result[1][3].x += reg_a[0].w * reg_b[1].x;
            result[1][3].y += reg_a[0].w * reg_b[1].y;
            result[1][3].z += reg_a[0].w * reg_b[1].z;
            result[1][3].w += reg_a[0].w * reg_b[1].w;

            // ----------------------

            result[2][0].x += reg_a[1].x * reg_b[0].x;
            result[2][0].y += reg_a[1].x * reg_b[0].y;
            result[2][0].z += reg_a[1].x * reg_b[0].z;
            result[2][0].w += reg_a[1].x * reg_b[0].w;
            result[2][1].x += reg_a[1].y * reg_b[0].x;
            result[2][1].y += reg_a[1].y * reg_b[0].y;
            result[2][1].z += reg_a[1].y * reg_b[0].z;
            result[2][1].w += reg_a[1].y * reg_b[0].w;
            result[2][2].x += reg_a[1].z * reg_b[0].x;
            result[2][2].y += reg_a[1].z * reg_b[0].y;
            result[2][2].z += reg_a[1].z * reg_b[0].z;
            result[2][2].w += reg_a[1].z * reg_b[0].w;
            result[2][3].x += reg_a[1].w * reg_b[0].x;
            result[2][3].y += reg_a[1].w * reg_b[0].y;
            result[2][3].z += reg_a[1].w * reg_b[0].z;
            result[2][3].w += reg_a[1].w * reg_b[0].w;

            // ----------------------

            result[3][0].x += reg_a[1].x * reg_b[1].x;
            result[3][0].y += reg_a[1].x * reg_b[1].y;
            result[3][0].z += reg_a[1].x * reg_b[1].z;
            result[3][0].w += reg_a[1].x * reg_b[1].w;
            result[3][1].x += reg_a[1].y * reg_b[1].x;
            result[3][1].y += reg_a[1].y * reg_b[1].y;
            result[3][1].z += reg_a[1].y * reg_b[1].z;
            result[3][1].w += reg_a[1].y * reg_b[1].w;
            result[3][2].x += reg_a[1].z * reg_b[1].x;
            result[3][2].y += reg_a[1].z * reg_b[1].y;
            result[3][2].z += reg_a[1].z * reg_b[1].z;
            result[3][2].w += reg_a[1].z * reg_b[1].w;
            result[3][3].x += reg_a[1].w * reg_b[1].x;
            result[3][3].y += reg_a[1].w * reg_b[1].y;
            result[3][3].z += reg_a[1].w * reg_b[1].z;
            result[3][3].w += reg_a[1].w * reg_b[1].w;
        }

        if( tileIdx < cTileK )
        {
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4    ) * TILE_M + ty_4 ) = ldA[0].x;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 1) * TILE_M + ty_4 ) = ldA[0].y;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 2) * TILE_M + ty_4 ) = ldA[0].z;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 3) * TILE_M + ty_4 ) = ldA[0].w;

            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4    ) * TILE_M + ty_4 + 64) = ldA[1].x;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 1) * TILE_M + ty_4 + 64) = ldA[1].y;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 2) * TILE_M + ty_4 + 64) = ldA[1].z;
            *((float*)&smemA[write_stage_idx][0][0] + (tx_4*4 + 3) * TILE_M + ty_4 + 64) = ldA[1].w;

            smemB[write_stage_idx][ty_32  ][tx_32] = ldB[0];
            smemB[write_stage_idx][ty_32+8][tx_32] = ldB[1];
            write_stage_idx ^= 1;
            __syncthreads();
        }

    }while(tileIdx < cTileK);

#pragma unroll
    for(int i=0; i<4; ++i)
    {
#pragma unroll
        for(int j=0; j<4; ++j)
        {
            result[i][j].x *= alpha;
            result[i][j].y *= alpha;
            result[i][j].z *= alpha;
            result[i][j].w *= alpha;
        }
    }
    float *startPointer[4] = {
            C + (blockIdx.y * TILE_M + threadIdx.y * 4     ) * ldc + blockIdx.x * TILE_N + threadIdx.x * 4,
            C + (blockIdx.y * TILE_M + threadIdx.y * 4     ) * ldc + blockIdx.x * TILE_N + threadIdx.x * 4 + 64,
            C + (blockIdx.y * TILE_M + threadIdx.y * 4 + 64) * ldc + blockIdx.x * TILE_N + threadIdx.x * 4,
            C + (blockIdx.y * TILE_M + threadIdx.y * 4 + 64) * ldc + blockIdx.x * TILE_N + threadIdx.x * 4 + 64
    };
#pragma unroll
    for( int i=0; i<4; ++i)
    {
#pragma unroll
        for( int j=0; j<4; ++j)
        {
            if( beta == 0 ){
                *(float4*)( startPointer[i] + j * ldc ) = result[i][j];
            }else{
                float4 value = *(float4*)( startPointer[i] + j * ldc );
                value.x = beta * value.x + result[i][j].x;
                value.y = beta * value.y + result[i][j].y;
                value.z = beta * value.z + result[i][j].z;
                value.w = beta * value.w + result[i][j].w;
                *(float4*)( startPointer[i] + j * ldc ) = value;
            }
        }
    }
}

void MySgemm( hipblasOperation_t transa,
              hipblasOperation_t transb,
              int m,
              int n,
              int k,
              float alpha, /* host or device pointer */
              const float* A,
              int lda,
              const float* B,
              int ldb,
              float beta, /* host or device pointer */
              float* C,
              int ldc)
{
    // currently only support row major input
    assert(transa == HIPBLAS_OP_T);
    assert(transb == HIPBLAS_OP_T);
    assert(m % 128 == 0);
    assert(n % 128 == 0);
    assert(k % 128 == 0);

    dim3 dimBlock(16, 16);
    dim3 dimGrid(n/TILE_N, m/TILE_M);
    if( n % TILE_N != 0 ) dimGrid.x += 1;
    if( m % TILE_M != 0 ) dimGrid.y += 1;
    int cTileK = k / TILE_K;
    if( k % TILE_K != 0 ) cTileK++;
    MatrixMul_T_T<<<dimGrid, dimBlock>>>(cTileK, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
}

int main() {
    const int m = 2048;
    const int k = 1024;
    const int n = 2048;
    const int iter_count = 500;
    cv::Mat matA(m, k, CV_32FC1);
    cv::randn(matA, 0.5, 1);
    cv::Mat matB(k, n, CV_32FC1);
    cv::randn(matB, 0.5, 1);
    auto cpuT1 = std::chrono::high_resolution_clock::now();
    cv::Mat matC = matA * matB;
    auto cpuT2 = std::chrono::high_resolution_clock::now();
    std::cout << std::chrono::duration_cast<std::chrono::microseconds>(cpuT2 - cpuT1).count() / 1000000.0 << std::endl;
    cv::Mat matD(n, m, CV_32FC1);
    cv::Mat matE(m, n, CV_32FC1);

    auto pMatA = MyCudaMalloc(m*k*sizeof(float));
    hipMemcpy(pMatA.get(), (void*)matA.data, matA.rows * matA.cols * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
    auto pMatB = MyCudaMalloc(k*n*sizeof(float));
    hipMemcpy(pMatB.get(), (void*)matB.data, matB.rows * matB.cols * sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);
    auto pMatC = MyCudaMalloc(m*n*sizeof(float));
    auto pMatD = MyCudaMalloc(m*n*sizeof(float));
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float alpha = 1, beta = 0;
    double total = 0;
    for( int i=0; i<iter_count; ++i)
    {
        hipEvent_t evtStart, evtFinish;
        hipEventCreate(&evtStart);
        hipEventCreate(&evtFinish);
        hipMemset(pMatC.get(), 0, m*n*sizeof(float));
        hipEventRecord(evtStart, hipStreamDefault);
        hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, &alpha, (float*)pMatA.get(), k, (float*)pMatB.get(), n, &beta, (float*)pMatC.get(), m );
        hipEventRecord(evtFinish, hipStreamDefault);
        hipEventSynchronize(evtFinish);
        float ms = 0;
        hipEventElapsedTime(&ms, evtStart, evtFinish);
        total += ms;
        hipEventDestroy(evtStart);
        hipEventDestroy(evtFinish);
    }
    hipMemcpy(matD.data, pMatC.get(), m*n*sizeof(float), hipMemcpyDeviceToHost);
    matD = matD.t();
    std::cout << total / 1000.0 / iter_count  << std::endl;

    total = 0;
    for( int i=0; i<iter_count; ++i)
    {
        hipEvent_t evtStart, evtFinish;
        hipEventCreate(&evtStart);
        hipEventCreate(&evtFinish);
        hipMemset(pMatD.get(), 0, m*n*sizeof(float));
        hipEventRecord(evtStart, hipStreamDefault);
        MySgemm(HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, alpha, (float*)pMatA.get(), k, (float*)pMatB.get(), n, beta, (float*)pMatD.get(), n );
        hipEventRecord(evtFinish, hipStreamDefault);
        hipEventSynchronize(evtFinish);
        float ms = 0;
        hipEventElapsedTime(&ms, evtStart, evtFinish);
        total += ms;
        hipEventDestroy(evtStart);
        hipEventDestroy(evtFinish);
    }
    std::cout << total / 1000.0 / iter_count << std::endl;

    hipMemcpy(matE.data, pMatD.get(), m*n*sizeof(float), hipMemcpyDeviceToHost);

    for(int i=0; i<m; ++i)
    {
        std::cout << matD.at<float>(0, i) << "\t" << matE.at<float>(0, i) << "\t" << matC.at<float>(0, i) << std::endl;
    }

//    cv::imshow("test", (matE - matC)*500);
//    cv::waitKey(0);

    return 0;
}
